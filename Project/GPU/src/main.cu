#include "hip/hip_runtime.h"
// Project
#include <util.cuh>

// std
#include <fstream>

__global__ void render(datatype *fb, int max_x, int max_y) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= max_x || y >= max_y) {
        return;
    }
    int pixel_index = y * max_x * 3 + x * 3;
    fb[pixel_index + 0] = datatype(x) / max_x;
    fb[pixel_index + 1] = datatype(y) / max_y;
    fb[pixel_index + 2] = 0.2;
}

int main() {
    // output setup
    std::ofstream output("image.ppm");
    std::streambuf* standard_out = std::cout.rdbuf();
    std::cout.rdbuf(output.rdbuf());

    int image_width          = 1920;
    int image_height         = 1080;
    int num_pixels           = image_width * image_height;
    size_t frame_buffer_size = 3 * num_pixels * sizeof(datatype);

    datatype* frame_buffer;
    checkCudaErrors(hipMallocManaged((void**) &frame_buffer, frame_buffer_size));

    dim3 db(TPB.x, TPB.y);
    dim3 dg((image_width + db.x - 1) / db.x, (image_height + db.y - 1) / db.y);
    render<<<dg, db>>>(frame_buffer, image_width, image_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Output FB as Image
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = 0; j < image_height; j++) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * 3 * image_width + i * 3;
            datatype r = frame_buffer[pixel_index + 0];
            datatype g = frame_buffer[pixel_index + 1];
            datatype b = frame_buffer[pixel_index + 2];
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    checkCudaErrors(hipFree(frame_buffer));

    // restore stdout
    std::cout.rdbuf(standard_out);
    output.close();
    return 0;
}